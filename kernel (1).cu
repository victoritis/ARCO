﻿// includes 
#include <stdio.h> 
#include <stdlib.h> 
#include <time.h> 
#include <hip/hip_runtime.h> 
 
// declaracion de funciones 
// GLOBAL: funcion llamada desde el host y ejecutada en el device (kernel)
__global__ void suma_GPU(int *dev_A,int *dev_B,int *dev_resultado,int n)
{
	int myID = threadIdx.x;
	dev_B[myID] = dev_A[(n-1)-myID];
	dev_resultado[myID] = dev_A[myID] + dev_B[myID];
}
__host__ void propiedades_Device(int deviceID)
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, deviceID);
	// calculo del numero de cores (SP)
	int cudaCores = 0;
	int SM = deviceProp.multiProcessorCount;
	int major = deviceProp.major;
	int minor = deviceProp.minor;
	const char* archName;
	switch (major)
	{
	case 1:
		//TESLA 
		archName = "TESLA";
		cudaCores = 8;
		break;
	case 2:
		//FERMI
		archName = "FERMI";
		if (minor == 0)
			cudaCores = 32;
		else
			cudaCores = 48;
		break;
	case 3:
		//KEPLER
		archName = "KEPLER";
		cudaCores = 192;
		break;
	case 5:
		//MAXWELL
		archName = "MAXWELL";
		cudaCores = 128;
		break;
	case 6:
		//PASCAL
		archName = "PASCAL";
		cudaCores = 64;
		break;
	case 7:
		//VOLTA(7.0) //TURING(7.5) 
		cudaCores = 64;
		if (minor == 0)
			archName = "VOLTA";
		else
			archName = "TURING";
		break;
	case 8:
		// AMPERE
		archName = "AMPERE";
		cudaCores = 64;
		break;
	default:
		//ARQUITECTURA DESCONOCIDA
		archName = "DESCONOCIDA";
	}
	int rtV;
	hipRuntimeGetVersion(&rtV);
	// presentacion de propiedades
	printf("***************************************************\n");
	printf("DEVICE %d: %s\n", deviceID, deviceProp.name);
	printf("***************************************************\n");
	printf("> CUDA Toolkit \t\t\t: %d.%d\n", rtV / 1000, (rtV % 1000) / 10);
	printf("> Arquitectura CUDA \t\t: %s\n", archName);
	printf("> Capacidad de Computo \t\t: %d.%d\n", major, minor);
	printf("> No. MultiProcesadores \t: %d\n", SM);
	printf("> No. Nucleos CUDA (%dx%d) \t: %d\n", cudaCores, SM, cudaCores* SM);
	printf("> Memoria Global (total) \t: %u MiB\n",
		deviceProp.totalGlobalMem / (1024 * 1024));
	printf("***************************************************\n");
}
__host__ int maximoHilos(int deviceID)
{
	
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, deviceID);
	int maxThreads = deviceProp.maxThreadsPerBlock;
	return maxThreads;
}

// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv)
{
	// declaraciones
	int* hst_salida1, * hst_salida2, * hst_resultado;
	int* dev_salida1, * dev_salida2, * dev_resultado;
	int n = 0;
	int maximo = maximoHilos(0);
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0)
	{
		printf("!!!!!No se han encontrado dispositivos CUDA!!!!!\n");
		printf("<pulsa [INTRO] para finalizar>");
		getchar();
		return 1;
	}
	else
	{
		printf("Se han encontrado <%d> dispositivos CUDA:\n", deviceCount);
		for (int id = 0; id < deviceCount; id++)
		{
			propiedades_Device(id);
		}
	}
	do{
		printf("Introduzca el numero de elementos: ");
		scanf("%d", &n);
		if(n > maximo)
			printf("\nERROR: numero maximo de hilos superado! [%d hilos]\n", maximo);
	}while (n>maximo);
	// reserva en el host
	hst_salida1 = (int*)malloc(n * sizeof(int));
	hst_salida2 = (int*)malloc(n * sizeof(int));
	hst_resultado = (int*)malloc(n * sizeof(int));
	// reserva en el device
	hipMalloc((void**)&dev_salida1, n * sizeof(int));
	hipMalloc((void**)&dev_salida2, n * sizeof(int));
	hipMalloc((void**)&dev_resultado, n * sizeof(int));

	srand((int)time(NULL));
	for (int i = 0; i < n; i++)
	{
		hst_salida1[i] = rand() % 9;
		hst_salida2[i] = 0;
	}

	hipMemcpy(dev_salida1, hst_salida1, n * sizeof(int), hipMemcpyHostToDevice);
	// EJECUCIÓN EN EL DEVICE
	// llamada a la funcion "impares_GPU()"
	suma_GPU<<< 1,n >>>(dev_salida1, dev_salida2, dev_resultado,n);

	// recogida de datos desde el device

	hipMemcpy(hst_salida2, dev_salida2, n * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(hst_resultado, dev_resultado, n * sizeof(int), hipMemcpyDeviceToHost);
	// impresion de resultados GPU
	printf("> SALIDA DE LA GPU:\n");
	printf("VECTOR 1:\n");
	for (int i = 0; i < n; i++)
		printf(" %2d", hst_salida1[i]);
	printf("\nVECTOR 2:\n");
	for (int i = 0; i < n; i++)
		printf(" %2d", hst_salida2[i]);
	printf("\nSUMA:\n");
	for (int i = 0; i < n; i++)
		printf(" %2d", hst_resultado[i]);
	printf("\n\n");
	
	hipFree(dev_salida1);
	hipFree(dev_salida2);
	hipFree(dev_resultado);

	time_t fecha;
	time(&fecha);
	printf("***************************************************\n");
	printf("Programa ejecutado el: %s\n", ctime(&fecha));
	printf("<pulsa [INTRO] para finalizar>");
	getchar();
	return 0;
}