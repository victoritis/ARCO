///////////////////////////////////////////////////////////////////////////
// PROGRAMACI�N EN CUDA C/C++
// Curso Basico
// PRACTICA 3: "HILOS Y BLOQUES"
//
// SEPTIEMBRE 2021
///////////////////////////////////////////////////////////////////////////
// includes
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

///////////////////////////////////////////////////////////////////////////
// defines
#define KIBI (1<<10) // KiB = 2^10
#define MEBI (1<<20) // MiB = 2^20
#define HB 5 //cada bloque va a tener 10 hilos como m�ximo


///////////////////////////////////////////////////////////////////////////
// Prototipos de funciones
int elementosVector(int);

//////////////////////////////////////////////////////////////////////////
// FUNCION GLOBAL QUE TRABAJA CON EL DEVICE
__global__ void suma(int* vector1, int* vector2, int* resultado, int n) {
	//identificador de hilo
	int myID = threadIdx.x + blockDim.x * blockIdx.x ;
	if (myID<n){
		vector2[myID] = vector1[n - 1 - myID];//obtenemos el vector 2 invirtiendo el vector 1
		resultado[myID] = vector1[myID] + vector2[myID];//sumamos los dos vectores y escribimos el resultado
	}	
}
/////////////////////////////////////////////////////////////////////////

//pedida de elementos del vector
int elementosVector(int MaxThreads) {
	int num;
	do {
		printf("Introduce el numero de elementos de los vectores a sumar \n");
		scanf("%d", &num);
		if (num > MaxThreads){
			printf("El numero de elementos introducido es mayor que el numero de hilos.\n");
		}
	} while (num > MaxThreads);
	getchar();// capturamos un INTRO para que no se cierre la consola de MSVS
	printf("\n");
	return num;
}

// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv)
{
	// buscando dispositivos
	int deviceCount;
	int MaxThreads;
	int n;
	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0)
	{
		printf("!!!!!No se han encontrado dispositivos CUDA!!!!!\n");
		printf("<pulsa [INTRO] para finalizar>");
		getchar();
		return 1;
	}
	else
	{
		printf("Se han encontrado <%d> dispositivos CUDA:\n", deviceCount);
		for (int deviceID = 0; deviceID < deviceCount; deviceID++)
		{
			hipDeviceProp_t deviceProp;
			hipGetDeviceProperties(&deviceProp, deviceID);
			// calculo del numero de cores (SP)
			int cudaCores = 0;
			int SM = deviceProp.multiProcessorCount;
			int major = deviceProp.major;
			int minor = deviceProp.minor;
			MaxThreads = deviceProp.maxThreadsPerBlock;
			switch (major)
			{
			case 1:
				//TESLA
				cudaCores = 8;
				break;
			case 2:
				//FERMI
				if (minor == 0)
					cudaCores = 32;
				else
					cudaCores = 48;
				break;
			case 3:
				//KEPLER
				cudaCores = 192;
				break;
			case 5:
				//MAXWELL
				cudaCores = 128;
				break;
			case 6:
				//PASCAL
				cudaCores = 64;
				break;
			case 7:
				//VOLTA (7.0) TURING (7.5)
				cudaCores = 64;
				break;
			case 8:
				//AMPERE
				cudaCores = 64;
				break;
			default:
				//ARQUITECTURA DESCONOCIDA
				cudaCores = 0;
				printf("!!!!!dispositivo desconocido!!!!!\n");
			}
			// presentacion de propiedades
			printf("*****************\n");
			printf("DEVICE %d: %s\n", deviceID, deviceProp.name);
			printf("*****************\n");
			printf("> Capacidad de Computo            \t: %d.%d\n", major, minor);
			printf("> No. de MultiProcesadores        \t: %d \n", SM);
			printf("> No. de CUDA Cores (%dx%d)       \t: %d \n", cudaCores, SM, cudaCores * SM);
			printf("> Memoria Global (total)          \t: %zu MiB\n", deviceProp.totalGlobalMem / MEBI);
			printf("> Memoria Compartida (por bloque) \t: %zu KiB\n", deviceProp.sharedMemPerBlock / KIBI);
			printf("> Memoria Constante  (total)      \t: %zu KiB\n", deviceProp.totalConstMem / KIBI);
			printf("  Numero maximo de hilos (por bloque) \t: %d\n", deviceProp.maxThreadsPerBlock);
			printf(" [x -> %d]\n [y -> %d]\n [z -> %d]\n",deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
			printf("> Numero maximo de bloques (por eje)\n");
			printf(" [x -> %d]\n [y -> %d]\n [z -> %d]\n",deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
			printf("*****************\n");
		}
	}
	n=elementosVector(MaxThreads);

	// declaraciones
	int* hst_vector;
	int* dev_vector1, * dev_vector2, * dev_resultado;

	// reserva en el host
	hst_vector = (int*)malloc(n * sizeof(int));

	// reserva en el device 
	hipMalloc((void**)&dev_vector1, n * sizeof(int));
	hipMalloc((void**)&dev_vector2, n * sizeof(int));
	hipMalloc((void**)& dev_resultado, n * sizeof(int));


	// creamos el primer vector en el host y el segundo en el device
	srand((int)time(NULL));
	for (int i = 0; i < n; i++)
	{
		hst_vector[i] = rand() % 10;
	}

	// copia de datos CPU -> GPU 
	hipMemcpy(dev_vector1, hst_vector, n * sizeof(int), hipMemcpyHostToDevice);

	//lanzamiento del Kernel
	int num_bloques = n/HB; //n=numero de elementos, HB=numero hilos por bloque
	if (n % HB !=0){
		num_bloques=num_bloques + 1;
	}
	printf("Vector de %d elementos \n", n);
	printf("Lanzamiento con %d bloques de %d hilos (%d hilos) \n", num_bloques, HB, num_bloques*HB);
	suma <<<num_bloques,HB>>> (dev_vector1, dev_vector2, dev_resultado, n);

	//recogida de datos desde el device (GPU -> CPU) e impresion de resultados
	hipMemcpy(hst_vector, dev_vector1, n * sizeof(int), hipMemcpyDeviceToHost);
	printf("\nVECTOR 1: \n");
	for (int i = 0; i < n; i++) {
		printf("%2d ", hst_vector[i]);
	}
	printf("\n");

	hipMemcpy(hst_vector, dev_vector2, n * sizeof(int), hipMemcpyDeviceToHost);
	printf("\nVECTOR 2: \n");
	for (int i = 0; i < n; i++) {
		printf("%2d ", hst_vector[i]);
	}
	printf("\n");

	hipMemcpy(hst_vector, dev_resultado, n * sizeof(int), hipMemcpyDeviceToHost);
	printf("\nVECTOR RESULTADO: \n");
	for (int i = 0; i < n; i++) {
		printf("%2d ", hst_vector[i]);
	}

	// salida del programa
	time_t fecha;
	time(&fecha);
	printf("\n*****************\n");
	printf("\nPrograma ejecutado el: %s\n", ctime(&fecha));

	// capturamos un INTRO para que no se cierre la consola de MSVS
	printf("<pulsa [INTRO] para finalizar>");
	getchar();
	return 0;
}
///////////////////////////////////////////////////////////////////////////
